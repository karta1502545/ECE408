#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <iostream>
using namespace std;

// reference: https://github.com/aschuh703/ECE408/tree/main/Project#milestone-2-baseline-convolutional-kernel

#define TILE_WIDTH 16
// tileWidth=28, ((3+(28-1)*4)^2+3*3)*4 = 49320 > 49152(maxShareMemorySize per block), shared memory overflow!
// tileWidth=27, ((3+(27-1)*4)^2+3*3)*4 = 45832 > 49152(maxShareMemorySize per block), time = 10.5+60.1=70.6ms
// tileWidth=16, time = 8.6 + 43.6 = 52.2ms

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int inSize = K + (TILE_WIDTH-1) * S;
    // const int inSize = TILE_WIDTH + K - 1;
    const int maskSize = K * K;
    const int in2dSize = inSize * inSize;

    // extern __shared__ float in2d[];
    // extern __shared__ float mask2d[];
    extern __shared__ float sharedMem[];

    // float *in2d = sharedMem;
    // float *mask2d = &sharedMem[in2dSize];
    float *mask2d = &sharedMem[0];
    float *in2d = &sharedMem[maskSize];

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    #define mask_2d(i1, i0) mask2d[(i1) * (K) + (i0)]
    #define in_2d(i1, i0) in2d[(i1) * (inSize) + (i0)]

    const int W_Grid = ceil(float(W_out) / TILE_WIDTH); // IMPORTANT: for grid, we see an output element as a block unit.
    int b = blockIdx.x;
    int m = blockIdx.z;
    int h = (blockIdx.y / W_Grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_Grid) * TILE_WIDTH + threadIdx.x;

    int hBase = (blockIdx.y / W_Grid) * TILE_WIDTH * S;
    int wBase = (blockIdx.y % W_Grid) * TILE_WIDTH * S;
    int ty = threadIdx.y, tx = threadIdx.x;

    // Insert your GPU convolution kernel code here
    float pValue = 0.0;
    for (int c=0; c<C; c++) {
        // load into shared memory
        if (ty < K && tx < K)
            mask_2d(ty, tx) = mask_4d(m, c, ty, tx);

        for (int i=ty; i<inSize; i+=blockDim.y) {
            for (int j=tx; j<inSize; j+=blockDim.x) {
                if (hBase+i < H && wBase+j < W)
                    in_2d(i, j) = in_4d(b, c, hBase+i, wBase+j);
                else
                    in_2d(i, j) = 0.0;
            }
        }
        
        __syncthreads();
        for (int kr=0; kr<K; kr++) {
            for (int kc=0; kc<K; kc++) {
                pValue += in_2d(ty*S+kr, tx*S+kc) * mask_2d(kr, kc);
            }
        }
        __syncthreads();
    }

    if (h < H_out && w < W_out) {
        out_4d(b, m, h, w) = pValue;
    }


    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    hipMalloc((void**) device_output_ptr, H_out * W_out * M * B * sizeof(float));
    hipMalloc((void**) device_input_ptr, H * W * C * B * sizeof(float));
    hipMalloc((void**) device_mask_ptr, M * C * K * K * sizeof(float));

    // IMPORTANT: Pointer level should be right.
    hipMemcpy(*device_input_ptr, host_input, H * W * C * B * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);
    // get_device_properties();

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    // const int H_out = (H)/S + 1;
    // const int W_out = (W)/S + 1;

    // const int H_out = H;
    // const int W_out = W;

    const int H_Grid = ceil(H_out / float(TILE_WIDTH));
    const int W_Grid = ceil(W_out / float(TILE_WIDTH));
    const int G = H_Grid*W_Grid;
    // const int G = ceil(H / float(TILE_WIDTH)) * ceil(W / float(TILE_WIDTH));
    // cout << "H_out = " << H_out << ", W_out = " << W_out << endl;
    // cout << "H_Grid = " << H_Grid << ", W_Grid = " << W_Grid << endl;
    // cout << "G = " << G << endl;

    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 DimGrid(B, G, M); // batch_size, GridSize, # of mask
    // int inSize = TILE_WIDTH * S + K - 1;
    // const int inSize = TILE_WIDTH + K - 1;
    const int inSize = K + (TILE_WIDTH-1) * S;
    size_t sharedMemSize = inSize * inSize * sizeof(float) + K * K * sizeof(float);

    conv_forward_kernel<<<DimGrid,DimBlock,sharedMemSize>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    hipDeviceSynchronize(); // not sure if I need this.
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    // Copy the output back to host
    hipMemcpy(host_output, device_output, H_out * W_out * M * B * sizeof(float), hipMemcpyDeviceToHost);
    // for (int i=0; i<H_out * W_out; i++) {
    //     cout << host_output[i] << " ";
    // }
    // cout << endl;
   
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
