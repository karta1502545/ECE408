#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <iostream>
using namespace std;

// reference: https://github.com/aschuh703/ECE408/tree/main/Project#milestone-2-baseline-convolutional-kernel

__constant__ float MASK_c[10000];
#define TILE_WIDTH 16
// tileWidth=28, ((3+(28-1)*4)^2+3*3)*4 = 49320 > 49152(maxShareMemorySize per block), shared memory overflow!
// tileWidth=27, ((3+(27-1)*4)^2+3*3)*4 = 45832 > 49152(maxShareMemorySize per block), time = 10.5+60.1=70.6ms
// tileWidth=16, time = 8.6 + 43.6 = 52.2ms

// with constant memory, time = 7.83 + 37.42 = 45.25ms

/*
Conv-GPU==
Layer Time: 297.666 ms
Op Time: 6.69915 ms
Conv-GPU==
Layer Time: 246.629 ms
Op Time: 33.0789 ms

Conv-GPU==
Layer Time: 319.333 ms
Op Time: 6.62453 ms
Conv-GPU==
Layer Time: 257.743 ms
Op Time: 32.8471 ms

Conv-GPU==
Layer Time: 302.691 ms
Op Time: 6.7325 ms
Conv-GPU==
Layer Time: 252.77 ms
Op Time: 33.0872 ms
*/

__global__ void conv_forward_kernel(float * __restrict__ output, const float * __restrict__ input, const float * __restrict__ mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int inSize = K + (TILE_WIDTH-1) * S;
    // const int inSize = TILE_WIDTH + K - 1;
    const int maskSize = K * K;
    const int in2dSize = inSize * inSize;

    extern __shared__ float sharedMem[];

    float *in2d = &sharedMem[0];

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) MASK_c[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    // #define mask_2d(i1, i0) mask2d[(i1) * (K) + (i0)]
    #define in_2d(i1, i0) in2d[(i1) * (inSize) + (i0)]

    const int W_Grid = ceil(float(W_out) / TILE_WIDTH); // IMPORTANT: for grid, we see an output element as a block unit.
    int b = blockIdx.x;
    int m = blockIdx.z;
    int h = (blockIdx.y / W_Grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.y % W_Grid) * TILE_WIDTH + threadIdx.x;

    int hBase = (blockIdx.y / W_Grid) * TILE_WIDTH * S;
    int wBase = (blockIdx.y % W_Grid) * TILE_WIDTH * S;
    
    // int ty = threadIdx.y, tx = threadIdx.x;
    // Insert your GPU convolution kernel code here
    float pValue[4];

    for (int ty=2*threadIdx.y; ty<2*(threadIdx.y+1); ty++) {
        for (int tx=2*threadIdx.x; tx<2*(threadIdx.x+1); tx++) {
            int py = (ty-2*threadIdx.y), px = (tx-2*threadIdx.x);
            int pIdx = py*2 + px;
            pValue[pIdx] = 0.0;
            if (h+py < H_out && w+px < W_out) {
                for (int c=0; c<C; c++) {
                    // load into shared memory
                    // if (ty < K && tx < K)
                        // mask_2d(ty, tx) = mask_4d(m, c, ty, tx);

                    for (int k=ty; k<inSize; k+=blockDim.y) {
                        for (int l=tx; l<inSize; l+=blockDim.x) {
                            if (hBase+k < H && wBase+l < W)
                                in_2d(k, l) = in_4d(b, c, hBase+k, wBase+l);
                            else
                                in_2d(k, l) = 0.0;
                        }
                    }
                    
                    __syncthreads();
                    for (int kr=0; kr<K; kr+=2) {
                        for (int kc=0; kc<K; kc+=2) {
                            /* 2 by 2 */
                            if (kr < K && kc < K)
                                pValue[pIdx] += in_2d(ty*S+kr, tx*S+kc) * mask_4d(m, c, kr, kc);
                            if (kr < K && kc+1 < K)
                                pValue[pIdx] += in_2d(ty*S+kr, tx*S+kc+1) * mask_4d(m, c, kr, kc+1);
                            if (kr+1 < K && kc < K)
                                pValue[pIdx] += in_2d(ty*S+kr+1, tx*S+kc) * mask_4d(m, c, kr+1, kc);
                            if (kr+1 < K && kc+1 < K)
                                pValue[pIdx] += in_2d(ty*S+kr+1, tx*S+kc+1) * mask_4d(m, c, kr+1, kc+1);
                        }
                    }
                    __syncthreads();
                }
            }
        }
    }

    for (int i=0; i<2; i++) 
        for (int j=0; j<2; j++)
            if (h+i < H_out && w+j < W_out)
                out_4d(b, m, h+i, w+j) = pValue[i*2+j];

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    hipMalloc((void**) device_output_ptr, H_out * W_out * M * B * sizeof(float));
    hipMalloc((void**) device_input_ptr, H * W * C * B * sizeof(float));
    // hipMalloc((void**) device_mask_ptr, M * C * K * K * sizeof(float));

    // IMPORTANT: Pointer level should be right.
    hipMemcpy(*device_input_ptr, host_input, H * W * C * B * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(*device_mask_ptr, host_mask, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(MASK_c), host_mask, M*C*K*K*sizeof(float));
    // get_device_properties();

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    // const int H_out = (H)/S + 1;
    // const int W_out = (W)/S + 1;

    // const int H_out = H;
    // const int W_out = W;

    const int H_Grid = ceil(H_out / float(TILE_WIDTH));
    const int W_Grid = ceil(W_out / float(TILE_WIDTH));
    const int G = H_Grid*W_Grid;
    // const int G = ceil(H / float(TILE_WIDTH)) * ceil(W / float(TILE_WIDTH));
    // cout << "H_out = " << H_out << ", W_out = " << W_out << endl;
    // cout << "H_Grid = " << H_Grid << ", W_Grid = " << W_Grid << endl;
    // cout << "G = " << G << endl;

    dim3 DimBlock(TILE_WIDTH/2, TILE_WIDTH/2, 1);
    dim3 DimGrid(B, G, M); // batch_size, GridSize, # of mask
    // int inSize = TILE_WIDTH * S + K - 1;
    // const int inSize = TILE_WIDTH + K - 1;
    const int inSize = K + (TILE_WIDTH-1) * S;
    size_t sharedMemSize = inSize * inSize * sizeof(float);

    conv_forward_kernel<<<DimGrid,DimBlock,sharedMemSize>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    hipDeviceSynchronize(); // not sure if I need this.
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    // Copy the output back to host
    hipMemcpy(host_output, device_output, H_out * W_out * M * B * sizeof(float), hipMemcpyDeviceToHost);
    // for (int i=0; i<H_out * W_out; i++) {
    //     cout << host_output[i] << " ";
    // }
    // cout << endl;
   
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    // hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
