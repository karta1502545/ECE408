#include "hip/hip_runtime.h"
#include <wb.h>
// #include <iostream>
// using namespace std;
#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void spmvJDSKernel(float *out, int *matColStart, int *matCols,
                              int *matRowPerm, int *matRows,
                              float *matData, float *vec, int dim, int sizeOfMatColStart) { 
  //@@ insert spmv kernel for jds format
  // dim => num_rows, out => y, vec => x
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  if (row < dim) {
    float dot = 0;
    unsigned int sec = 0;
    while (matRows[row] > sec) {
    // while (sec+1 < sizeOfMatColStart && (matColStart[sec+1] - matColStart[sec] > row)) { // this cannot work since we do not have the last index of sec (in lec20 p25, we would only have [0,3,6], not [0,3,6,7])
      dot += matData[matColStart[sec]+row] * vec[matCols[matColStart[sec]+row]];
      sec += 1;
    }
    out[matRowPerm[row]] = dot;
  }
}

static void spmvJDS(float *out, int *matColStart, int *matCols,
                    int *matRowPerm, int *matRows, float *matData,
                    float *vec, int dim, int sizeOfMatColStart) {

  //@@ invoke spmv kernel for jds format
  // dim maybe larger than 1024, grid size should be ceil(dim * 1.0 / BlockSize)
  // BlockSize can be 1024?
  // why we need matRows?
  dim3 DimGrid(ceil(dim / float(BLOCK_SIZE)), 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);
  spmvJDSKernel<<<DimGrid, DimBlock>>>(out, matColStart, matCols, matRowPerm, matRows, matData, vec, dim, sizeOfMatColStart);
}

int main(int argc, char **argv) {
  wbArg_t args;
  int *hostCSRCols;
  int *hostCSRRows;
  float *hostCSRData;
  int *hostJDSColStart;
  int *hostJDSCols;
  int *hostJDSRowPerm;
  int *hostJDSRows;
  float *hostJDSData;
  float *hostVector;
  float *hostOutput;
  int *deviceJDSColStart;
  int *deviceJDSCols;
  int *deviceJDSRowPerm;
  int *deviceJDSRows;
  float *deviceJDSData;
  float *deviceVector;
  float *deviceOutput;
  int dim, ncols, nrows, ndata;
  int maxRowNNZ;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostCSRCols = (int *)wbImport(wbArg_getInputFile(args, 0), &ncols, "Integer");
  hostCSRRows = (int *)wbImport(wbArg_getInputFile(args, 1), &nrows, "Integer");
  hostCSRData = (float *)wbImport(wbArg_getInputFile(args, 2), &ndata, "Real");
  hostVector = (float *)wbImport(wbArg_getInputFile(args, 3), &dim, "Real");

  hostOutput = (float *)malloc(sizeof(float) * dim);

  wbTime_stop(Generic, "Importing data and creating memory on host");

  CSRToJDS(dim, hostCSRRows, hostCSRCols, hostCSRData, &hostJDSRowPerm, &hostJDSRows,
           &hostJDSColStart, &hostJDSCols, &hostJDSData);
  maxRowNNZ = hostJDSRows[0];

  wbTime_start(GPU, "Allocating GPU memory.");
  hipMalloc((void **)&deviceJDSColStart, sizeof(int) * maxRowNNZ);
  hipMalloc((void **)&deviceJDSCols, sizeof(int) * ndata);
  hipMalloc((void **)&deviceJDSRowPerm, sizeof(int) * dim);
  hipMalloc((void **)&deviceJDSRows, sizeof(int) * dim);
  hipMalloc((void **)&deviceJDSData, sizeof(float) * ndata);

  hipMalloc((void **)&deviceVector, sizeof(float) * dim);
  hipMalloc((void **)&deviceOutput, sizeof(float) * dim);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  hipMemcpy(deviceJDSColStart, hostJDSColStart, sizeof(int) * maxRowNNZ,
             hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSCols, hostJDSCols, sizeof(int) * ndata, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSRowPerm, hostJDSRowPerm, sizeof(int) * dim, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSRows, hostJDSRows, sizeof(int) * dim, hipMemcpyHostToDevice);
  hipMemcpy(deviceJDSData, hostJDSData, sizeof(float) * ndata, hipMemcpyHostToDevice);
  hipMemcpy(deviceVector, hostVector, sizeof(float) * dim, hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  wbTime_start(Compute, "Performing CUDA computation");
  spmvJDS(deviceOutput, deviceJDSColStart, deviceJDSCols, deviceJDSRowPerm, deviceJDSRows,
          deviceJDSData, deviceVector, dim, maxRowNNZ);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  hipMemcpy(hostOutput, deviceOutput, sizeof(float) * dim, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceVector);
  hipFree(deviceOutput);
  hipFree(deviceJDSColStart);
  hipFree(deviceJDSCols);
  hipFree(deviceJDSRowPerm);
  hipFree(deviceJDSRows);
  hipFree(deviceJDSData);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, dim);
  // cout << maxRowNNZ << endl;
  // for (int i=0; i<maxRowNNZ; i++) {
  //   cout << hostJDSColStart[i] << " ";
  // }
  // cout << endl;
  // for (int i=0; i<dim; i++) {
  //   cout << hostJDSRows[i] << " ";
  // }
  // cout << endl;
  free(hostCSRCols);
  free(hostCSRRows);
  free(hostCSRData);
  free(hostVector);
  free(hostOutput);
  free(hostJDSColStart);
  free(hostJDSCols);
  free(hostJDSRowPerm);
  free(hostJDSRows);
  free(hostJDSData);

  return 0;
}
